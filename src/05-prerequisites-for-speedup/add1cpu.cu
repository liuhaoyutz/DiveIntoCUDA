#include "hip/hip_runtime.h"
// 这段代码演示了怎样使用CUDA提供的基于CUDA事件（CUDA event）的计时，计算函数add使用CPU，而没有使用GPU
/*
执行结果如下：
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/05-prerequisites-for-speedup$ nvcc add1cpu.cu 
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/05-prerequisites-for-speedup$ ./a.out 
Time = 493.389 ms.
Time = 309.624 ms.
Time = 313.205 ms.
Time = 312.053 ms.
Time = 309.701 ms.
Time = 312.842 ms.
Time = 320.212 ms.
Time = 310.194 ms.
Time = 317.703 ms.
Time = 320.36 ms.
Time = 319.866 ms.
Time = 314.576 +- 4.26743 ms.
No errors
*/


#include "error.cuh"
#include <math.h>
#include <stdio.h>

#ifdef USE_DP  // 使用双精度
    typedef double real;
    const real EPSILON = 1.0e-15;
#else  // 使用单精度
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 10;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;
void add(const real *x, const real *y, real *z, const int N);
void check(const real *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(real) * N;

    // 给host数组分配内存
    real *x = (real*) malloc(M);
    real *y = (real*) malloc(M);
    real *z = (real*) malloc(M);

    // 初始化host数组
    for (int n = 0; n < N; ++n)
    {
        x[n] = a;
        y[n] = b;
    }

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        // 创建CUDA event变量start和stop。
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        // 将start传递给hipEventRecord，在需要计时的代码块之前，记录一个代表开始的事件。
        CHECK(hipEventRecord(start));
        /*
        对处于 TCC 驱动模式的 GPU 来说可以省略，但对处于 WDDM 驱动模式的 GPU 来说必须保留。
        这是因为，在处于 WDDM 驱动模式的 GPU 中，一个 CUDA 流（CUDA stream）中的操作
        （如这里的 hipEventRecord 函数）并不是直接提交给 GPU 执行，而是先提交到一个软件队列，
        需要添加一条对该流的 hipEventQuery 操作（或者 hipEventSynchronize）刷新队列，才能促使前面的操作在 GPU 执行。
        */
        hipEventQuery(start);

        // 执行计算，add是CPU函数
        add(x, y, z, N);

        // 将stop传递给hipEventRecord。在需要计时的代码块之后记录一个代表结束的事件。
        CHECK(hipEventRecord(stop));
        // 调用hipEventSynchronize 函数让主机等待事件 stop 被记录完毕。
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        //调用 hipEventElapsedTime 函数计算 start 和 stop 之间的时间差(单位ms)。
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        // 调用 hipEventDestroy 函数销毁 start 和 stop 这两个 CUDA 事件。
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    check(z, N);

    free(x);
    free(y);
    free(z);
    return 0;
}

void add(const real *x, const real *y, real *z, const int N)
{
    for (int n = 0; n < N; ++n)
    {
        z[n] = x[n] + y[n];
    }
}

void check(const real *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}


