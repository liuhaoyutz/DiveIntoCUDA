#include "hip/hip_runtime.h"
// 这段代码展示了memcpy用时远大于计算用时

/*
执行结果如下：
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/05-prerequisites-for-speedup$ nvcc add3memcpy.cu 
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/05-prerequisites-for-speedup$ ./a.out 
Time = 342.539 ms.
Time = 139.054 ms.
Time = 139.005 ms.
Time = 138.968 ms.
Time = 138.302 ms.
Time = 138.272 ms.
Time = 138.29 ms.
Time = 140.278 ms.
Time = 138.604 ms.
Time = 138.561 ms.
Time = 138.289 ms.
Time = 138.762 +- 0.586302 ms.
No errors
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/05-prerequisites-for-speedup$ 
*/

#include "error.cuh"
#include <math.h>
#include <stdio.h>

#ifdef USE_DP  // 使用双精度浮点数
    typedef double real;
    const real EPSILON = 1.0e-15;
#else  // 使用单精度浮点数
    typedef float real;
    const real EPSILON = 1.0e-6f;
#endif

const int NUM_REPEATS = 10;
const real a = 1.23;
const real b = 2.34;
const real c = 3.57;
void __global__ add(const real *x, const real *y, real *z, const int N);
void check(const real *z, const int N);

int main(void)
{
    const int N = 100000000;
    const int M = sizeof(real) * N;
    // 为host数组分配内存
    real *h_x = (real*) malloc(M);
    real *h_y = (real*) malloc(M);
    real *h_z = (real*) malloc(M);

    // 初始化host数组
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = a;
        h_y[n] = b;
    }

    // 为device数组分配显存
    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void **)&d_x, M));
    CHECK(hipMalloc((void **)&d_y, M));
    CHECK(hipMalloc((void **)&d_z, M));

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        // 创建CUDA event变量start和stop。
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        // 将start传递给hipEventRecord，在需要计时的代码块之前，记录一个代表开始的事件。
        CHECK(hipEventRecord(start));
        /*
        对处于 TCC 驱动模式的 GPU 来说可以省略，但对处于 WDDM 驱动模式的 GPU 来说必须保留。
        这是因为，在处于 WDDM 驱动模式的 GPU 中，一个 CUDA 流（CUDA stream）中的操作
        （如这里的 hipEventRecord 函数）并不是直接提交给 GPU 执行，而是先提交到一个软件队列，
        需要添加一条对该流的 hipEventQuery 操作（或者 hipEventSynchronize）刷新队列，才能促使前面的操作在 GPU 执行。
        */
        hipEventQuery(start);

        // 将host数组内容拷贝到device数组
        CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));
        // 调用核函数
        add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
        // 将结果从device拷贝到host
        CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));

        // 将stop传递给hipEventRecord。在需要计时的代码块之后记录一个代表结束的事件。
        CHECK(hipEventRecord(stop));
        // 调用hipEventSynchronize 函数让主机等待事件 stop 被记录完毕。
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        //调用 hipEventElapsedTime 函数计算 start 和 stop 之间的时间差(单位ms)。
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        // 调用 hipEventDestroy 函数销毁 start 和 stop 这两个 CUDA 事件。
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    return 0;
}

void __global__ add(const real *x, const real *y, real *z, const int N)
{
    // 计算每个线程对应的index
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    // 实际上我们创建的线程个数是N + block_size - 1，只允许前N个线程执行计算，后面的线程不做任何计算。
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void check(const real *z, const int N)
{
    bool has_error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

