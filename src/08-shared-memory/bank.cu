#include "hip/hip_runtime.h"
// 这段代码是一个基于CUDA的矩阵转置示例，主要展示了两种不同的共享内存实现方式（有和没有共享内存银行冲突）对性能的影响。

#include "error.cuh"
#include <stdio.h>

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 10;
const int TILE_DIM = 32;

void timing(const real *d_A, real *d_B, const int N, const int task);
__global__ void transpose1(const real *A, real *B, const int N);
__global__ void transpose2(const real *A, real *B, const int N);
void print_matrix(const int N, const real *A);

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("usage: %s N\n", argv[0]);
        exit(1);
    }
    const int N = atoi(argv[1]);

    const int N2 = N * N;
    const int M = sizeof(real) * N2;
    real *h_A = (real *) malloc(M);
    real *h_B = (real *) malloc(M);
    for (int n = 0; n < N2; ++n)
    {
        h_A[n] = n;
    }
    real *d_A, *d_B;
    CHECK(hipMalloc(&d_A, M));
    CHECK(hipMalloc(&d_B, M));
    CHECK(hipMemcpy(d_A, h_A, M, hipMemcpyHostToDevice));

    // 调用 timing 函数，分别测试两种转置实现的性能：
    // transpose1：存在共享内存bank冲突。
    // transpose2：优化以避免共享内存bank冲突。
    printf("\ntranspose with shared memory bank conflict:\n");
    timing(d_A, d_B, N, 1);
    printf("\ntranspose without shared memory bank conflict:\n");
    timing(d_A, d_B, N, 2);

    CHECK(hipMemcpy(h_B, d_B, M, hipMemcpyDeviceToHost));
    if (N <= 10)
    {
        printf("A =\n");
        print_matrix(N, h_A);
        printf("\nB =\n");
        print_matrix(N, h_B);
    }

    free(h_A);
    free(h_B);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}

void timing(const real *d_A, real *d_B, const int N, const int task)
{
    const int grid_size_x = (N + TILE_DIM - 1) / TILE_DIM;
    const int grid_size_y = grid_size_x;
    const dim3 block_size(TILE_DIM, TILE_DIM);
    const dim3 grid_size(grid_size_x, grid_size_y);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        switch (task)
        {
            case 1:
                transpose1<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            case 2:
                transpose2<<<grid_size, block_size>>>(d_A, d_B, N);
                break;
            default:
                printf("Error: wrong task\n");
                exit(1);
                break;
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
}

__global__ void transpose1(const real *A, real *B, const int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM];
    int bx = blockIdx.x * TILE_DIM;
    int by = blockIdx.y * TILE_DIM;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    if (nx1 < N && ny1 < N)
    {
        S[threadIdx.y][threadIdx.x] = A[ny1 * N + nx1];
    }
    __syncthreads();

    int nx2 = bx + threadIdx.y;
    int ny2 = by + threadIdx.x;
    if (nx2 < N && ny2 < N)
    {
        B[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}

__global__ void transpose2(const real *A, real *B, const int N)
{
    __shared__ real S[TILE_DIM][TILE_DIM + 1];
    int bx = blockIdx.x * TILE_DIM;
    int by = blockIdx.y * TILE_DIM;

    int nx1 = bx + threadIdx.x;
    int ny1 = by + threadIdx.y;
    if (nx1 < N && ny1 < N)
    {
        S[threadIdx.y][threadIdx.x] = A[ny1 * N + nx1];
    }
    __syncthreads();

    int nx2 = bx + threadIdx.y;
    int ny2 = by + threadIdx.x;
    if (nx2 < N && ny2 < N)
    {
        B[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}

void print_matrix(const int N, const real *A)
{
    for (int ny = 0; ny < N; ny++)
    {
        for (int nx = 0; nx < N; nx++)
        {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}

/*
执行结果如下：

(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/08-shared-memory$ nvcc bank.cu 
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/08-shared-memory$ ./a.out 8

transpose with shared memory bank conflict:
Time = 0.191488 ms.
Time = 0.007168 ms.
Time = 0.00512 ms.
Time = 0.005984 ms.
Time = 0.004928 ms.
Time = 0.004096 ms.
Time = 0.0048 ms.
Time = 0.004096 ms.
Time = 0.003296 ms.
Time = 0.004096 ms.
Time = 0.004928 ms.
Time = 0.0048512 +- 0.00104245 ms.

transpose without shared memory bank conflict:
Time = 0.017408 ms.
Time = 0.00512 ms.
Time = 0.004096 ms.
Time = 0.004096 ms.
Time = 0.00512 ms.
Time = 0.003808 ms.
Time = 0.004096 ms.
Time = 0.004096 ms.
Time = 0.004064 ms.
Time = 0.005824 ms.
Time = 0.003968 ms.
Time = 0.0044288 +- 0.000638432 ms.
A =
0	1	2	3	4	5	6	7	
8	9	10	11	12	13	14	15	
16	17	18	19	20	21	22	23	
24	25	26	27	28	29	30	31	
32	33	34	35	36	37	38	39	
40	41	42	43	44	45	46	47	
48	49	50	51	52	53	54	55	
56	57	58	59	60	61	62	63	

B =
0	8	16	24	32	40	48	56	
1	9	17	25	33	41	49	57	
2	10	18	26	34	42	50	58	
3	11	19	27	35	43	51	59	
4	12	20	28	36	44	52	60	
5	13	21	29	37	45	53	61	
6	14	22	30	38	46	54	62	
7	15	23	31	39	47	55	63	
(INT8) haoyu@thinker:~/work/code/DiveIntoCUDA/src/08-shared-memory$ 
*/