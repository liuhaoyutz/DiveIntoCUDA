#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

const unsigned WIDTH = 8;
const unsigned BLOCK_SIZE = 16;
const unsigned FULL_MASK = 0xffffffff;

void __global__ test_warp_primitives(void);

int main(int argc, char **argv)
{
    test_warp_primitives<<<1, BLOCK_SIZE>>>();
    CHECK(hipDeviceSynchronize());
    return 0;
}

void __global__ test_warp_primitives(void)
{
    int tid = threadIdx.x;
    int lane_id = tid % WIDTH;

    if (tid == 0) printf("threadIdx.x: ");
    printf("%2d ", tid);
    if (tid == 0) printf("\n");

    if (tid == 0) printf("lane_id:     ");
    printf("%2d ", lane_id);
    if (tid == 0) printf("\n");

    unsigned mask1 = __ballot_sync(FULL_MASK, tid > 0);  // 除了第1个线程不满足tid>0，对应位设置为0，其他都满足，所以mask1为fffe
    unsigned mask2 = __ballot_sync(FULL_MASK, tid == 0); // 除了第1个线程满足tid==0，对应位设置为1，其他都不满足，所以mask2为1
    if (tid == 0) printf("FULL_MASK = %x\n", FULL_MASK); // 打印ffffffff
    if (tid == 1) printf("mask1     = %x\n", mask1);     // 打印fffe
    if (tid == 0) printf("mask2     = %x\n", mask2);     // 打印1

    int result = __all_sync(FULL_MASK, tid);  // 第1个线程tid为0，所以result为0
    if (tid == 0) printf("all_sync (FULL_MASK): %d\n", result);

    result = __all_sync(mask1, tid);  // 第1个线程不参与，所以result为1
    if (tid == 1) printf("all_sync     (mask1): %d\n", result);

    result = __any_sync(FULL_MASK, tid);
    if (tid == 0) printf("any_sync (FULL_MASK): %d\n", result);

    result = __any_sync(mask2, tid);
    if (tid == 0) printf("any_sync     (mask2): %d\n", result);

    int value = __shfl_sync(FULL_MASK, tid, 2, WIDTH);
    if (tid == 0) printf("shfl:      ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_up_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_up:   ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_down_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_down: ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_xor_sync(FULL_MASK, tid, 1, WIDTH);
    if (tid == 0) printf("shfl_xor:  ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");
}

